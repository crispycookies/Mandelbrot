#include "hip/hip_runtime.h"
#include <iostream>
#include "kernel.cuh"
#include "math.h"

__device__ auto global_thread_idx_x() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ float norm(hipFloatComplex & z)
{
    auto x = z.x * z.x;
    auto y = z.y * z.y;
    return sqrt(x+y);
}


__device__
int iterate(const hipFloatComplex & c) noexcept {
    auto i {0};
    hipFloatComplex z = {0};
    do{
        z = hipCmulf(z,z);
        z = hipCaddf(z,c);
    }while((i++ < g_colors) && ((int)norm(z)<g_infinity));



    return i;
}


__global__ void iterate_GPU(pfc::pixel_t * gpu_ptr, float xright, float xleft, float yright, float yleft, const float x_zp, const float y_zp, int height, int width, const size_t count, float factor) {
    size_t const current_idx = global_thread_idx_x();
    size_t const iteration_idx = current_idx/(height*width);

    for(size_t i = 0; i < iteration_idx; i++){
        xright -= (xright - x_zp) * (1-factor);
        yright -= (yright - y_zp) * (1-factor);
        xleft -= (xleft - x_zp) * (1-factor);
        yleft -= (yleft - y_zp) * (1-factor);
    }

    float dx = (xright - xleft)/(float)(width - 1);
    float dy = (yright - yleft)/(float)(height - 1);

    int x = (int)current_idx % width;
    int y = (int)current_idx / width - (int)(iteration_idx*width);

    hipComplex c;
    c.x = {xleft + ((float)x)*dx};
    c.y = {yright - (float)y*dy};

    if (current_idx < (height * width * count)) {
        gpu_ptr[current_idx] = {pfc::byte_t(iterate(c)),0,0};
    }
    //printf("%f ,%f, %f, %f\n", xright, yright, xleft, yleft);

}


hipError_t call_iteration_kernel(pfc::pixel_t * gpu_ptr, std::complex<float> & left, std::complex<float>  & right, const std::complex<float>  & zPoint, int height, int width, float factor, const size_t count){

    auto const size{ static_cast <int> (height*width) * count};

    auto const  tib =
            1024;

    auto xleft = left.real();
    auto yleft = left.imag();
    auto xright = right.real();
    auto yright = right.imag();
/*
    xright -= (xright - zPoint.real()) * (1-factor);
    yright -= (yright - zPoint.imag()) * (1-factor);
    xleft -= (xleft - zPoint.real()) * (1-factor);
    yleft -= (yleft - zPoint.imag()) * (1-factor);
*/
    iterate_GPU <<<((size+tib-1)/tib),tib >>> (gpu_ptr,  xright, xleft, yright, yleft, zPoint.real(), zPoint.imag(),height, width,count, factor);

    for(size_t i = 0; i < count; i++){
        xright -= (xright - zPoint.real()) * (1-factor);
        yright -= (yright - zPoint.imag()) * (1-factor);
        xleft -= (xleft - zPoint.real()) * (1-factor);
        yleft -= (yleft - zPoint.imag()) * (1-factor);
    }

    left = {xleft, yleft};
    right = {xright, yright};


    std::cout << "LEFT:" <<left << std::endl;
    std::cout << "RIGHT:" <<right << std::endl;

    hipDeviceSynchronize();
    return hipGetLastError();
}