#include "hip/hip_runtime.h"
#include <iostream>
#include "kernel.cuh"
#include "math.h"
#include "misc/pfc_threading.h"

__constant__ const int height = 4608;
__constant__ const int width = 8192;

__device__ auto global_thread_idx_x() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ inline float norm(hipFloatComplex & z)
{
    auto x = z.x * z.x;
    auto y = z.y * z.y;
    return x+y;
}


__device__
int iterate(const hipFloatComplex & c) noexcept {
    auto i {0};
    hipFloatComplex z = {0};
#pragma unroll
    do{
        z = hipCmulf(z,z);
        z = hipCaddf(z,c);
    }while((i++ < g_colors) && ((int)norm(z)<g_infinity));
    return i;
}


__global__ void iterate_GPU(pfc::pixel_t * gpu_ptr, float xright, float xleft, float yright, float yleft, int offset, float  dx, float dy) {
    size_t const current_idx = (global_thread_idx_x());

    int x = (int)(current_idx) % width;
    int y = (int)(current_idx) / width;

    hipComplex c;
    c.x = {xleft + ((float)x)*dx};
    c.y = {yright - (float)y*dy};

    if (current_idx < height*width) {
        gpu_ptr[current_idx] = {pfc::byte_t(iterate(c)),0,0};
    }
}


hipError_t call_iteration_kernel(pfc::pixel_t * gpu_ptr, std::complex<float> left, std::complex<float>  right, const std::complex<float>  & zPoint, int height, int width, float factor, hipStream_t * streams, int count){
    auto const size{ static_cast <int> (height*width) };

    auto const  tib = 512;

    auto xleft = left.real();
    auto yleft = left.imag();
    auto xright = right.real();
    auto yright = right.imag();

    for(int i = 1; i <= count; i++){
        xright -= (xright - zPoint.real()) * (1-factor);
        yright -= (yright - zPoint.imag()) * (1-factor);
        xleft -= (xleft - zPoint.real()) * (1-factor);
        yleft -= (yleft - zPoint.imag()) * (1-factor);
    }


    float dx = (xright - xleft)/(float)(width - 1);
    float dy = (yright - yleft)/(float)(height - 1);

    /*for(int i = 0; i < num_stream; i++){
        auto offset =  (size/num_stream)*(i);
        iterate_GPU <<<((size+tib-1)/(tib*num_stream)),tib ,0, streams[i]>>> (&gpu_ptr[offset],  xright, xleft, yright, yleft, height, width, offset, dx, dy);
    }*/
    auto offset =  gpu_ptr;
    iterate_GPU <<<((size+tib-1)/(tib)),tib ,0, *streams>>> (gpu_ptr,  xright, xleft, yright, yleft, 0 , dx, dy);


    return hipGetLastError();
}