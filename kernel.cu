#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "math.h"
#include "misc/pfc_threading.h"

__device__ auto global_thread_idx_x() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__device__ float norm(hipFloatComplex & z)
{
    auto x = z.x * z.x;
    auto y = z.y * z.y;
    return sqrt(x+y);
}


__device__
int iterate(const hipFloatComplex & c) noexcept {
    auto i {0};
    hipFloatComplex z = {0};
#pragma unroll
    do{
        z = hipCmulf(z,z);
        z = hipCaddf(z,c);
    }while((i++ < g_colors) && ((int)norm(z)<g_infinity));
    return i;
}


__global__ void iterate_GPU(pfc::pixel_t * gpu_ptr, float xright, float xleft, float yright, float yleft, int height, int width, int offset, float  dx, float dy) {
    size_t const current_idx = (global_thread_idx_x());

    int x = (int)(current_idx+offset) % width;
    int y = (int)(current_idx+offset) / width;

    hipComplex c;
    c.x = {xleft + ((float)x)*dx};
    c.y = {yright - (float)y*dy};

    if (current_idx < height*width) {
        gpu_ptr[current_idx] = {pfc::byte_t(iterate(c)),0,0};
        //gpu_ptr[current_idx+1] = {pfc::byte_t(iterate(c)),0,0};
    }
}


hipError_t call_iteration_kernel(pfc::pixel_t * gpu_ptr, std::complex<float> & left, std::complex<float>  & right, const std::complex<float>  & zPoint, int height, int width, float factor, hipStream_t * streams, int num_stream){

    auto const size{ static_cast <int> (height*width) };

    auto const  tib = 512;

    auto xleft = left.real();
    auto yleft = left.imag();
    auto xright = right.real();
    auto yright = right.imag();

    xright -= (xright - zPoint.real()) * (1-factor);
    yright -= (yright - zPoint.imag()) * (1-factor);
    xleft -= (xleft - zPoint.real()) * (1-factor);
    yleft -= (yleft - zPoint.imag()) * (1-factor);

    float dx = (xright - xleft)/(float)(width - 1);
    float dy = (yright - yleft)/(float)(height - 1);

    for(int i = 0; i < num_stream; i++){
        auto offset =  (size/num_stream)*(i);
        iterate_GPU <<<((size+tib-1)/(tib*num_stream)),tib ,0, streams[i]>>> (&gpu_ptr[offset],  xright, xleft, yright, yleft, height, width, offset, dx, dy);
    }

    //iterate_GPU <<<((size+tib-1)/(tib)),tib ,0>>> (gpu_ptr,  xright, xleft, yright, yleft, height, width, 0);

    left = {xleft, yleft};
    right = {xright, yright};

    hipDeviceSynchronize();
    return hipGetLastError();
}